#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// run matrix multiplication sequentially 

const static int N = 100;

// kernel function
__global__
void multMatrices(int* in1, int* in2, int* out, const int col, const int row) {
    int tmp = 0;
    for (int n = 0; n < N; n++) {
        tmp += in1[row * N + n] * in2[n * N + col];
    }
    out[row * N + col] = tmp;
}

int main() {
    int* in1, * in2, * out;
    int* Din1, * Din2, * Dout;

    int size = sizeof(float) * N * N;

    // allocate memory for host
    in1 = (int*)malloc(size);
    in2 = (int*)malloc(size);
    out = (int*)malloc(size);

    // assign random elemtens to both matrices
    for (int i = 0; i < N * N; i++) {
        in1[i] = rand() % 10;
    }
    for (int i = 0; i < N * N; i++) {
        in2[i] = rand() % 10;
    }

    // allocate memory for device
    hipMalloc((void**)&Din1, size);
    hipMalloc((void**)&Din2, size);
    hipMalloc((void**)&Dout, size);

    // copy input from host to device
    hipMemcpy(Din1, in1, size, hipMemcpyHostToDevice);
    hipMemcpy(Din2, in2, size, hipMemcpyHostToDevice);

    // iterate over all rows
    for (int row = 0; row < N; row++) {
        // iterate over all columns
        for (int col = 0; col < N; col++) {
            // calculate one element at a time
            multMatrices << <1, 1 >> > (Din1, Din2, Dout, col, row);
        }
    }

    // wait for GPU
    hipDeviceSynchronize();

    //copy result from device to host
    hipMemcpy(out, Dout, size, hipMemcpyDeviceToHost);

    for (int n = 0; n < N; n++) {
        for (int m = 0; m < N; m++) {
            printf("\t%d", out[n * N + m]);
        }
        printf("\n");
    }
    printf("\n");

    //free memory
    free(in1);
    free(in2);
    free(out);
    hipFree(Din1);
    hipFree(Din2);
    hipFree(Dout);

    return 0;
}