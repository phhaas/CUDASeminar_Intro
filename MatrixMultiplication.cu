﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


// run matrix multiplication with all calculations in parallel

const static int N = 100;

// kernel function
__global__
void multMatrices(int* in1, int* in2, int* out) {
    //global index
    int tmp = 0;
    for (int n = 0; n < blockDim.x; n++) {
        tmp += in1[threadIdx.x * blockDim.x + n] * in2[n * blockDim.x + blockIdx.x];
    }
    out[threadIdx.x * blockDim.x + blockIdx.x] = tmp;
}

int main() {
    int* in1, * in2, * out;
    int* Din1, * Din2, * Dout;

    int size    = sizeof(float) * N * N;

    // allocate memory for host
    in1 = (int*)malloc(size);
    in2 = (int*)malloc(size);
    out = (int*)malloc(size);

    // assign random elemtens to both matrices
    for (int i = 0; i < N * N; i++) {
        in1[i] = rand() % 10;
    }
    for (int i = 0; i < N * N; i++) {
        in2[i] = rand() % 10;
    }

    // allocate memory for device
    hipMalloc((void**)&Din1, size);
    hipMalloc((void**)&Din2, size);
    hipMalloc((void**)&Dout, size);

    // copy input from host to device
    hipMemcpy(Din1, in1, size, hipMemcpyHostToDevice);
    hipMemcpy(Din2, in2, size, hipMemcpyHostToDevice);

    // calculate addition for all entries in parallel
    multMatrices <<<N, N >>> (Din1, Din2, Dout);

    // wait for GPU
    hipDeviceSynchronize();

    //copy result from device to host
    hipMemcpy(out, Dout, size, hipMemcpyDeviceToHost);

    for (int n = 0; n < N; n++) {
        for (int m = 0; m < N; m++) {
            printf("\t%d", out[n*N + m]);
        }
        printf("\n");
    }
    printf("\n");

    //free memory
    free(in1);
    free(in2);
    free(out);
    hipFree(Din1);
    hipFree(Din2);
    hipFree(Dout);

    return 0;
}